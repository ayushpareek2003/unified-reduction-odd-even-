#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdio.h>

__global__ void reduction_sum_normal(float* values,float* sum, int len){
    // printf("at thread %d \n",threadIdx.x);
    int l=2*threadIdx.x;

    for(int st=1;st<len;st*=2){

        if(threadIdx.x % st==0 && l+st<len){
            printf("sorted %d and %d \n",l,l+st);
            values[l]+=values[l+st];
            
        }

        __syncthreads();

        
    }

    if(threadIdx.x==0){
        *sum=values[0];
    }
}


#define blockidm 37

__global__ void reduction_sum_shared(float* values,float* sum, int len){

    __shared__ float blockmem[blockidm];
    int i=threadIdx.x;
    if(blockidm+i<len){

        
        printf(" first go sorted %d and %d \n",i,i+blockidm);
        blockmem[i]=values[i]+values[i+blockidm];
        if(threadIdx.x==0 && blockidm!=1 && blockidm*2!=len){
            blockmem[0]+=values[blockidm-1];
        }

        int stride=blockidm/2;
        __syncthreads();
        while(stride>=1){

            

            if(threadIdx.x<stride){

                blockmem[i]=blockmem[i]+blockmem[i+stride];
                                printf(" sorted %d and %d  at ith stridde %d  updated value %f \n",i,i+stride,stride,blockmem[i]);
            }
            if(threadIdx.x==0 && stride!=1 &&(stride)%2!=0){

                
                blockmem[0]=blockmem[0]+blockmem[stride-1];
                printf("falted index adding %d to 0  value at %f \n",stride-1,blockmem[0]);
            }
            __syncthreads();
            stride=stride/2;

        }

        if(threadIdx.x==0){
            *sum=blockmem[0];
        }
    }
}


int main(){
    int a;
    std::cin>>a;

    float* memory;

    memory=(float*)malloc(a*sizeof(float));

    for(int i=0;i<a;i++){
        memory[i]=6785;
    }

    // std::cout<<memory[4]<<std::endl;

    float* dev;
    hipMalloc((void**)&dev,a*sizeof(float));
    hipMemcpy(dev,memory,a*sizeof(float),hipMemcpyHostToDevice);
    float* sum;

    hipMalloc((void**)&sum,sizeof(float));


    // reduction_sum_normal<<<(1),(a/2)+1>>>(dev,sum,a);

    reduction_sum_shared<<<(1),(a/2)+1>>>(dev,sum,a);


    float* ans=(float*)malloc(sizeof(float));
    hipMemcpy(ans,sum,sizeof(float),hipMemcpyDeviceToHost);
    std::cout<<"answer is :"<<*ans;
    return 0;



}

